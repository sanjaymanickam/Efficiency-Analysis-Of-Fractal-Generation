#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
  #include "bmp.h"
}

__global__ void render(char *img, int width, int height) {
  unsigned int x_dim = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y_dim = blockIdx.y*blockDim.y + threadIdx.y;
  int index = 3*width*y_dim + x_dim*3;
  float x_origin = ((float) x_dim/width)*3.25 - 2;
  float y_origin = ((float) y_dim/width)*2.5 - 1.25;

  float x = 0.0;
  float y = 0.0;

  int iteration = 0;
  int max_iteration = 256;
  while(x*x + y*y <= 4 && iteration < max_iteration) {
    float xtemp = x*x - y*y + x_origin;
    y = 2*x*y + y_origin;
    x = xtemp;
    iteration++;
  }

  if(iteration == max_iteration) {
    img[index] = 0;
    img[index + 1] = 0;
    img[index + 2] = 0;
  } else {
    img[index] = iteration;
    img[index + 1] = iteration;
    img[index + 2] = iteration;
  }
}

void runCUDA(int width, int height)
{
  size_t buffer_size = sizeof(char) * width * height * 3;

	char *image;
  hipMalloc((void **) &image, buffer_size);

  char *host_image = (char *) malloc(buffer_size);

  dim3 blockDim(16, 16, 1);
  dim3 gridDim(width / blockDim.x, height / blockDim.y, 1);
  render<<< gridDim, blockDim, 0 >>>(image, width, height);

  hipMemcpy(host_image, image, buffer_size, hipMemcpyDeviceToHost);

  write_bmp("output.bmp", width, height, host_image);

  hipFree(image);
  free(host_image);
}

int main(int argc, const char * argv[]) {
  runCUDA(1024, 1024);
  return 0;
}
